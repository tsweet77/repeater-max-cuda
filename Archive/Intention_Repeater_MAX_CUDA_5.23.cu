#include "hip/hip_runtime.h"
/*
    Intention Repeater MAX CUDA v5.23 (c)2020-2024 by Anthro Teacher aka Thomas Sweet.
    Updated 4/5/2024 by Anthro Teacher and Claude 3 Opus.
    To compile: nvcc -O3 Intention_Repeater_MAX_CUDA.cu -o Intention_Repeater_MAX_CUDA.exe -L/Users/tswee/miniconda3/Library/lib -lz
    Repeats your intention up to 100 PHz to make things happen.
    For help: intention_repeater_max_cuda.exe --help
    Intention Repeater MAX CUDA is powered by a Servitor (20 Years / 2000+ hours in the making) [HR 6819 Black Hole System].
    Servitor Info: https://enlightenedstates.com/2017/04/07/servitor-just-powerful-spiritual-tool/
    Website: https://www.intentionrepeater.com/
    Forum: https://forums.intentionrepeater.com/
    Licensed under GNU General Public License v3.0
    This means you can modify, redistribute and even sell your own modified software, as long as it's open source too and released under this same license.
    https://choosealicense.com/licenses/gpl-3.0/
*/

#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <cstring>
#include <ctime>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <iterator>
#include <locale.h>
#include <sstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <csignal>
#include <atomic>

std::atomic<bool> interrupted(false);

void signalHandler(int signum)
{
    //std::cout << "\nInterrupt signal (" << signum << ") received.\n";
    interrupted.store(true);
}

// CUDA kernel to perform intention repeating and frequency updating
__global__ void intentionRepeaterKernel(const char *intentionMultiplied, unsigned long long int *freq, size_t intentionSize)
{
    unsigned long long int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < intentionSize)
    {
        atomicAdd(freq, 1);
    }
}

#ifdef _WIN32
#include <windows.h>
#elif __linux__
#include <sys/sysinfo.h>
#elif __APPLE__
#include <sys/sysctl.h>
#endif

#include "picosha2.h"
#include "zlib.h"

constexpr int ONE_MINUTE = 60;
constexpr int ONE_HOUR = 3600;

#ifdef _WIN32
HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);

constexpr int BLACK = 0;
constexpr int BLUE = 1;
constexpr int GREEN = 2;
constexpr int CYAN = 3;
constexpr int RED = 4;
constexpr int MAGENTA = 5;
constexpr int YELLOW = 6;
constexpr int WHITE = 7;
constexpr int DARKGRAY = 8;
constexpr int LIGHTBLUE = 9;
constexpr int LIGHTGREEN = 10;
constexpr int LIGHTCYAN = 11;
constexpr int LIGHTRED = 12;
constexpr int LIGHTMAGENTA = 13;
constexpr int LIGHTYELLOW = 14;
constexpr int LIGHTGRAY = 15;

const char *enum2str[] = {"BLACK", "BLUE", "GREEN", "CYAN", "RED", "MAGENTA", "YELLOW", "WHITE", "DARKGRAY", "LIGHTBLUE", "LIGHTGREEN", "LIGHTCYAN", "LIGHTRED", "LIGHTMAGENTA", "LIGHTYELLOW", "LIGHTGRAY"};
#elif __APPLE__
#define DEFAULT "\033[0m"
#define DARKGRAY "\033[1;30m"
#define BLACK "\033[0;30m"
#define LIGHTRED "\033[1;31m"
#define RED "\033[0;31m"
#define LIGHTGREEN "\033[1;32m"
#define GREEN "\033[0;32m"
#define LIGHTYELLOW "\033[1;33m"
#define YELLOW "\033[0;33m"
#define LIGHTBLUE "\033[1;34m"
#define BLUE "\033[0;34m"
#define LIGHTMAGENTA "\033[1;35m"
#define MAGENTA "\033[0;35m"
#define LIGHTCYAN "\033[1;36m"
#define CYAN "\033[0;36m"
#define WHITE "\033[1;37m"
#define LIGHTGRAY "\033[0;37m"
#else
#define DEFAULT "\033[0m"
#define DARKGRAY "\033[1;30m"
#define BLACK "\033[0;30m"
#define LIGHTRED "\033[1;31m"
#define RED "\033[0;31m"
#define LIGHTGREEN "\033[1;32m"
#define GREEN "\033[0;32m"
#define LIGHTYELLOW "\033[1;33m"
#define YELLOW "\033[0;33m"
#define LIGHTBLUE "\033[1;34m"
#define BLUE "\033[0;34m"
#define LIGHTMAGENTA "\033[1;35m"
#define MAGENTA "\033[0;35m"
#define LIGHTCYAN "\033[1;36m"
#define CYAN "\033[0;36m"
#define WHITE "\033[1;37m"
#define LIGHTGRAY "\033[0;37m"
#endif

const std::string HSUPLINK_FILE = "HSUPLINK.TXT";

class comma_numpunct : public std::numpunct<char>
{
protected:
    virtual char do_thousands_sep() const
    {
        return ',';
    }

    virtual std::string do_grouping() const
    {
        return "\03";
    }
};

unsigned long long int get_ninety_percent_free_memory()
{
    unsigned long long int free_memory = 0;

#ifdef _WIN32
    // Windows-specific memory information
    MEMORYSTATUSEX memInfo;
    memInfo.dwLength = sizeof(MEMORYSTATUSEX);
    GlobalMemoryStatusEx(&memInfo);
    DWORDLONG freePhysMem = memInfo.ullAvailPhys;
    free_memory = static_cast<unsigned long long>(freePhysMem * 0.9); // 90% of free physical memory
#elif __linux__
    // Linux-specific memory information
    struct sysinfo memInfo;
    sysinfo(&memInfo);
    unsigned long long totalPhysMem = memInfo.totalram;
    totalPhysMem *= memInfo.mem_unit;
    unsigned long long freePhysMem = memInfo.freeram;
    freePhysMem *= memInfo.mem_unit;
    free_memory = static_cast<unsigned long long>(freePhysMem * 0.9); // 90% of free memory
#elif __APPLE__
    // macOS-specific memory information
    int mib[2];
    int64_t physical_memory;
    size_t length;

    mib[0] = CTL_HW;
    mib[1] = HW_MEMSIZE;
    length = sizeof(int64_t);
    sysctl(mib, 2, &physical_memory, &length, NULL, 0);

    unsigned long long totalPhysMem = static_cast<unsigned long long>(physical_memory);
    unsigned long long freePhysMem = totalPhysMem * 0.9; // 90% of total memory
    free_memory = freePhysMem;
#else
    std::cerr << "Unsupported operating system" << std::endl;
    return static_cast<unsigned long long>(-1); // Return max value to indicate error
#endif

    return free_memory;
}

std::string getHSUPLINKContents()
{
    std::string hsuplink;
    std::string HSUPLINK_FILE = "HSUPLINK.TXT";
    std::ifstream hsuplinkFile(HSUPLINK_FILE);

    if (hsuplinkFile.is_open())
    {
        std::ostringstream hsuplinkStream;
        hsuplinkStream << hsuplinkFile.rdbuf();
        hsuplink = hsuplinkStream.str();
        hsuplinkFile.close();
    }
    else
    {
        return HSUPLINK_FILE;
    }
    // std::cout << "HSUPLINK.TXT contents: " << hsuplink << std::endl;

    std::ifstream intentionsFile("INTENTIONS.TXT");
    if (intentionsFile.is_open())
    {
        std::ostringstream intentionsStream;
        intentionsStream << intentionsFile.rdbuf();
        std::string intentions = intentionsStream.str();
        intentionsFile.close();

        size_t pos = 0;
        while ((pos = hsuplink.find("INTENTIONS.TXT", pos)) != std::string::npos)
        {
            hsuplink.replace(pos, 14, intentions);
            pos += intentions.length();
        }
    }
    // std::cout << "HSUPLINK.TXT contents: " << hsuplink << std::endl;
    return hsuplink;
}

std::string compressMessage(const std::string &message)
{
    z_stream zs;
    std::memset(&zs, 0, sizeof(zs));

    if (deflateInit(&zs, Z_DEFAULT_COMPRESSION) != Z_OK)
    {
        return ""; // Compression initialization failed
    }

    zs.next_in = reinterpret_cast<Bytef *>(const_cast<char *>(message.data()));
    zs.avail_in = message.size();

    std::string compressed;
    char outbuffer[32768]; // Output buffer
    int ret;
    do
    {
        zs.next_out = reinterpret_cast<Bytef *>(outbuffer);
        zs.avail_out = sizeof(outbuffer);

        ret = deflate(&zs, Z_FINISH);

        if (compressed.size() < zs.total_out)
        {
            compressed.append(outbuffer, zs.total_out - compressed.size());
        }
    } while (ret == Z_OK);

    deflateEnd(&zs);

    if (ret != Z_STREAM_END)
    {
        return ""; // Compression failed
    }

    return compressed;
}

void readFileContents(const std::string &filename, std::string &intention_file_contents)
{
    std::ifstream file(filename, std::ios::binary);
    if (!file)
    {
        std::cerr << "File not found" << std::endl;
        std::exit(EXIT_FAILURE); // Terminate the program
    }

    std::ostringstream buffer;
    char ch;
    while (file.get(ch))
    {
        if (ch != '\0')
        {
            buffer.put(ch);
        }
    }

    intention_file_contents = buffer.str();
    file.close();
}

std::string display_suffix(std::string num, int power, std::string designator)
{
    if (power < 3)
    {
        return num;
    }

    std::string s;

    if (designator == "Iterations")
    {
        constexpr char iterations_suffix_array[] = {' ', 'k', 'M', 'B', 'T', 'q', 'Q', 's', 'S', 'O', 'N', 'D'};
        s = iterations_suffix_array[power / 3];
    }
    else // designator == "Frequency"
    {
        constexpr char frequency_suffix_array[] = {' ', 'k', 'M', 'G', 'T', 'P', 'E', 'Z', 'Y', 'R'};
        s = frequency_suffix_array[power / 3];
    }

    std::string str2 = num.substr(0, power % 3 + 1) + "." + num.substr(power % 3 + 1, 3) + s;

    return str2;
}

std::string FormatTimeRun(int seconds_elapsed)
{
    int hour, min, sec;
    std::string hour_formatted, min_formatted, sec_formatted;

    hour = seconds_elapsed / ONE_HOUR;
    seconds_elapsed -= hour * ONE_HOUR;
    min = seconds_elapsed / ONE_MINUTE;
    seconds_elapsed -= min * ONE_MINUTE;
    sec = seconds_elapsed;

    hour_formatted = (hour < 10) ? "0" + std::to_string(hour) : std::to_string(hour);
    min_formatted = (min < 10) ? "0" + std::to_string(min) : std::to_string(min);
    sec_formatted = (sec < 10) ? "0" + std::to_string(sec) : std::to_string(sec);

    return hour_formatted + ":" + min_formatted + ":" + sec_formatted;
}

void print_color_help()
{
#ifndef _WIN32
    std::cout << WHITE << "Color values for flag: --color [COLOR]" << std::endl
              << std::endl;

    std::cout << DEFAULT << "DEFAULT" << std::endl;
    std::cout << DARKGRAY << "DARKGRAY" << std::endl;
    std::cout << BLACK << "BLACK" << std::endl;
    std::cout << LIGHTRED << "LIGHTRED" << std::endl;
    std::cout << RED << "RED" << std::endl;
    std::cout << LIGHTGREEN << "LIGHTGREEN" << std::endl;
    std::cout << GREEN << "GREEN" << std::endl;
    std::cout << LIGHTYELLOW << "LIGHTYELLOW" << std::endl;
    std::cout << YELLOW << "YELLOW" << std::endl;
    std::cout << LIGHTBLUE << "LIGHTBLUE" << std::endl;
    std::cout << BLUE << "BLUE" << std::endl;
    std::cout << LIGHTMAGENTA << "LIGHTMAGENTA" << std::endl;
    std::cout << MAGENTA << "MAGENTA" << std::endl;
    std::cout << LIGHTCYAN << "LIGHTCYAN" << std::endl;
    std::cout << CYAN << "CYAN" << std::endl;
    std::cout << WHITE << "WHITE" << std::endl;
    std::cout << LIGHTGRAY << "LIGHTGRAY" << std::endl;
#else
    SetConsoleTextAttribute(hConsole, WHITE);
    std::cout << "Color values for flag: --color [COLOR]" << std::endl
              << std::endl;

    for (int k = 1; k <= 15; k++)
    {
        SetConsoleTextAttribute(hConsole, k);
        std::cout << enum2str[k] << std::endl;
    }
#endif // _WIN32
}

void create_nesting_files()
{
    std::ofstream myfile;

    myfile.open("NEST-1.TXT");
    if (myfile.is_open())
    {
        for (int repnum = 1; repnum <= 10; repnum++)
        {
            myfile << "INTENTIONS.TXT\r\n";
        }
    }
    myfile.close();

    for (int filenum = 2; filenum <= 100; filenum++)
    {
        std::string filename = "NEST-" + std::to_string(filenum) + ".TXT";
        myfile.open(filename);
        for (int repnum = 1; repnum <= 10; repnum++)
        {
            myfile << "NEST-" + std::to_string(filenum - 1) + ".TXT\r\n";
        }
        myfile.close();
    }

    std::cout << "Intention Repeater Nesting Files Written." << std::endl;
    std::cout << "Be sure to have your intentions in the INTENTIONS.TXT file." << std::endl;
    std::cout << "To run with the nesting option, use --boostlevel 50, for example to use up to Nesting to 50 levels deep." << std::endl;
    std::cout << "--boostlevel valid values: 1 to 100." << std::endl;
    std::cout << "When using --boostlevel 50, for example, it will ignore the --intent, and use \"NEST-50.TXT\" for the intent instead." << std::endl
              << std::endl;
}

void print_help()
{
    const std::string helpText = R"(
Intention Repeater MAX CUDA v5.23 (c)2020-2024 by Anthro Teacher aka Thomas Sweet.
This utility repeats your intention millions of times per second, in computer memory, to aid in manifestation.
Performance benchmark, exponents and flags by Karteek Sheri.
Holo-Link framework by Mystic Minds. This implementation by Anthro Teacher.
Intention multiplying by Anthro Teacher.

Optional Flags:
 a) --dur or -d, example: --dur 00:01:00
 b) --imem or -m, example: --imem 5
 c) --intent or -i, example: --intent "I am love."
 d) --suffix or -s, example: --suffix HZ
 e) --color or -c, example: --color LIGHTBLUE
 f) --boostlevel or -b, example: --boostlevel 100
 g) --createnestingfiles or -p, example: --createnestingfiles
 h) --usehololink or -u
 i) --createhololinkfiles or -x
 j) --colorhelp or -n
 k) --restevery or -e
 l) --restfor or -r
 m) --compress or -x
 n) --hashing or -g
 o) --file or -f
 p) --file2 or -f2
 q) --help or -h

--dur = Duration in HH:MM:SS format. Default = Run until stopped manually.
--imem = Specify how many GB of System RAM to use. Higher amount repeats faster, but takes longer to load. Default = 1.0.
--intent = Intention. Default = Prompts the user for intention.
--suffix = Specify whether to show regular (Hz) designator or scientific notation (Exp). Default = HZ.
--usehololink = Utilize the Holo-Link framework by Mystic Minds. 
--createhololinkfiles will create the default Holo-Link files and exit. Run with this option before using the --usehololink option. This will overwrite the TXT files.
--color = Set the text color. Default = WHITE.
--colorhelp = List and show all available colors for text using the option --color.
--createnestingfiles = Create the NEST- files required for boosting.
--boostlevel = Set the level to boost the power (1-100). Use --createnestingfiles before using --boostlevel.
--restevery = Stop repeating every specified # of seconds.
--restfor = # of Seconds to rest for each rest period.
--compress = Use compression Default n.
--hashing = Use hashing. Default n.
--file = Specify file to use if applicable.
--file2 = Specify second file to use if applicable.
--help = Display this help.

Example usage:
intention_repeater_max.exe --dur "00:01:00" --imem 4.0 --intent "I am Love." --hashing y --compress y

Example usage with Holo-Link:
1) intention_repeater_max.exe --createhololinkfiles
2) intention_repeater_max.exe --usehololink --color LIGHTBLUE --suffix EXP --dur 00:01:00 --imem 4.0 --intent "I am calm."

Make sure to create your INTENTIONS.TXT file, in this folder, with your intentions, before running #2 above.  
The --intent option is ignored when using --usehololink, which instead uses the INTENTIONS.TXT file.

Example usage with Nesting Files:
1) intention_repeater_max.exe --createnestingfiles
2) intention_repeater_max.exe --color LIGHTBLUE --dur 00:01:00 --imem 4.0 --boostlevel 5

Make sure to create your INTENTIONS.TXT file, in this folder, with your intentions, and the Nesting Files before running #2 above.

gitHub Repository: https://github.com/tsweet77/repeater-max 
Forum: https://forums.intentionrepeater.com
Website: https://www.intentionrepeater.com
)";

#ifndef _WIN32
    std::cout << helpText << std::endl;
#else
    SetConsoleTextAttribute(hConsole, WHITE);
    std::cout << helpText << std::endl;
    SetConsoleTextAttribute(hConsole, WHITE);
#endif
}

std::string getBoostIntention(std::string param_boostlevel)
{
    // Convert param_boostlevel to an integer
    int boostLevel = std::stoi(param_boostlevel);
    std::ostringstream NESTING_FILE_CONTENTS;

    // Check if boostLevel is within the valid range (1 to 100)
    if (boostLevel < 1 || boostLevel > 100)
    {
        return "0";
    }

    // Loop from 1 to param_boostlevel
    for (int i = 1; i <= boostLevel; i++)
    {
        std::string fileName = "NEST-" + std::to_string(i) + ".TXT";
        std::ifstream file1(fileName);

        if (!file1.is_open())
        {
            return "0";
        }

        NESTING_FILE_CONTENTS << file1.rdbuf();
        file1.close();

        std::ifstream file2("INTENTIONS.TXT");
        if (!file2.is_open())
        {
            return "0";
        }

        NESTING_FILE_CONTENTS << file2.rdbuf();
        file2.close();
    }

    return NESTING_FILE_CONTENTS.str();
}

std::string FindSum(std::string a, std::string b)
{
    std::vector<int> finalsum;
    int carry = 0;

    int i = a.size() - 1;
    int j = b.size() - 1;
    while ((i >= 0) && (j >= 0))
    {
        int x = (a[i] - '0') + (b[j] - '0') + carry;
        finalsum.push_back(x % 10);
        carry = x / 10;
        i--;
        j--;
    }

    while (i >= 0)
    {
        int x = (a[i] - '0') + carry;
        finalsum.push_back(x % 10);
        carry = x / 10;
        i--;
    }

    while (j >= 0)
    {
        int x = (b[j] - '0') + carry;
        finalsum.push_back(x % 10);
        carry = x / 10;
        j--;
    }

    while (carry)
    {
        finalsum.push_back(carry % 10);
        carry = carry / 10;
    }

    std::stringstream final_iter;
    std::copy(finalsum.rbegin(), finalsum.rend(), std::ostream_iterator<int>(final_iter, ""));

    return final_iter.str();
}

void create_hololink_files()
{
    const std::string HOLOSTONE_FILE = "HOLOSTONE.TXT";
    const std::string THOUGHTFORM_A_FILE = "THOUGHTFORM_A.TXT";
    const std::string THOUGHTFORM_B_FILE = "THOUGHTFORM_B.TXT";
    const std::string AMPLIFIER_FILE = "AMPLIFIER.TXT";

    std::ostringstream HOLOLINK_CONTENTS;
    HOLOLINK_CONTENTS << "#Comments are designated with a # prefix, and such commands are to be ignored by the Holo-Link.\r\n"
                      << "#" << HSUPLINK_FILE << " CONFIG FILE v1.0\r\n"
                      << "#Holo-Link framework created by Mystic Minds (2022).\r\n"
                      << "#This implementation of the Holo-Link framework by Anthro Teacher.\r\n"
                      << "\r\n"
                      << "DECLARATION PRIMARY (Properties of thought forms and uplink):\r\n"
                      << "\r\n"
                      << "I declare the uplink multiply the energy received from the Holo-Stones by Infinity and densify all energy to the highest amount to achieve Instant Quantum Manifestation of the energetic programmings in "
                      << HSUPLINK_FILE << ".\r\n"
                      << "\r\n"
                      << "I declare the Holo-Stones to funnel their energy into " << HOLOSTONE_FILE << ".\r\n"
                      << "\r\n"
                      << "I declare the Holo-Stones to amplify the power and receptivity of the energetic programmings in "
                      << HSUPLINK_FILE << ".\r\n"
                      << "\r\n"
                      << "I declare the Holo-Stones to multiply the strength of the energetic programmings in "
                      << HSUPLINK_FILE << " and increase the potency at the most optimal rate.\r\n"
                      << "\r\n"
                      << "I declare that all energetic programmings in " << HSUPLINK_FILE
                      << " be imprinted, imbued and amplified with the new energy from the Holo-Stones.\r\n"
                      << "\r\n"
                      << HOLOSTONE_FILE << ", " << AMPLIFIER_FILE << ", " << THOUGHTFORM_A_FILE << " AND "
                      << THOUGHTFORM_B_FILE
                      << " are extremely pure and of highest vibration and are fully optimized for Instant Quantum Manifestation.\r\n"
                      << "\r\n"
                      << THOUGHTFORM_A_FILE
                      << " is creating an unbreakable and continuous connection and funnel energy to all energetic programmings in "
                      << HSUPLINK_FILE << ".\r\n"
                      << "\r\n"
                      << THOUGHTFORM_A_FILE
                      << " uses energy from Infinite Source to continuously uphold a perfect link between the Holo-Stones and the "
                      << HSUPLINK_FILE
                      << " to bring in infinitely more energy into all energetic programmings in "
                      << HSUPLINK_FILE << ".\r\n"
                      << "\r\n"
                      << THOUGHTFORM_B_FILE << " reinforces 100% of energy into all the energetic programmings in "
                      << HSUPLINK_FILE << " at the quantum level.\r\n"
                      << "\r\n"
                      << THOUGHTFORM_B_FILE
                      << " safely and efficiently removes all blockages in this system at the quantum level to allow for Instant Quantum Manifestation.\r\n"
                      << "\r\n"
                      << HOLOSTONE_FILE << " feeds " << AMPLIFIER_FILE
                      << " which amplifies the energy and feeds it back to " << HOLOSTONE_FILE
                      << " and repeats it to the perfect intensity.\r\n"
                      << "\r\n"
                      << "All energetic programmings listed in " << HSUPLINK_FILE
                      << " are now amplified to the highest power, speed and quantum-level precision using energy from the Holo-Stones which are sourced through "
                      << HSUPLINK_FILE << ".\r\n"
                      << "\r\n"
                      << HOLOSTONE_FILE
                      << " works with Earth's Crystal Grid in the most optimal way possible for Instant Quantum Manifestation.\r\n"
                      << "\r\n"
                      << "Earth's Power Grid is extremely pure, cool, clean, efficient, optimized, and of highest vibration and is safely tapped in the most optimal way possible by HOLOSTONE.TXT for Instant Quantum Manifestation, and uses the least amount of electricity possible for everyone who desires this.\r\n"
                      << "UPLINK CORE (Reference any object, file, spell, etc. here):\r\n"
                      << "\r\n"
                      << HOLOSTONE_FILE
                      << " (Receives and distributes energy to all objects, files, spells, etc referenced below):\r\n"
                      << "\r\n"
                      << "[INSERT OBJECTS TO CHARGE]\r\n"
                      << "\r\n"
                      << "INTENTIONS.TXT\r\n"
                      << "\r\n"
                      << "DECLARATIONS SECONDARY (Add-ons that strengthen the properties of the uplink itself):\r\n"
                      << "\r\n"
                      << "I declare the Holo-Stones will uplink their energy into these energetic programmings in "
                      << HSUPLINK_FILE
                      << " to create instant, immediate and prominent results optimally, efficiently and effortlessly.\r\n"
                      << "\r\n"
                      << "I declare these energetic programmings in " << HSUPLINK_FILE
                      << " to grow stronger at the most optimal rate through the ever-growing power of the Holo-Stones.\r\n"
                      << "\r\n"
                      << "I call upon the Holo-Stones to channel the Atlantean Master Crystals, Infinite Source, Earth's Crystal Grid and Earth's Power Grid directly and utilize their energy as a funnel into HOLOSTONE.TXT which will then funnel into the energetic programmings in "
                      << HSUPLINK_FILE << ".\r\n"
                      << "\r\n"
                      << "The energetic programmings specified in " << HSUPLINK_FILE
                      << " are now being perfected and fully optimized.\r\n"
                      << "\r\n"
                      << "I declare that the more the energetic programmings in " << HSUPLINK_FILE
                      << " are used, the stronger they become.\r\n"
                      << "\r\n"
                      << "I am in my highest and most optimal reality/timeline.\r\n"
                      << "\r\n"
                      << "I am grounded, cleared, healed, balanced, strong-willed and I release what I do not need.\r\n"
                      << "\r\n"
                      << "Every day, in every way, it's getting better and better.\r\n"
                      << "\r\n"
                      << "The Atlantean Master Crystals AND Earth's Crystal Grid are open to Infinite Source.\r\n"
                      << "\r\n"
                      << "For my highest good and the highest good of all.\r\n"
                      << "\r\n"
                      << "Thank you. So be it. OM.\r\n"
                      << "ALL ABOVE STATEMENTS RESPECT THE FREE WILL OF ALL INVOLVED.\r\n";

    std::ofstream HOLOSTONE_FILE_FILE(HOLOSTONE_FILE);
    HOLOSTONE_FILE_FILE << "HOLOSTONE";
    std::ofstream THOUGHTFORM_A_FILE_FILE(THOUGHTFORM_A_FILE);
    THOUGHTFORM_A_FILE_FILE << "THOUGHTFORM A";
    std::ofstream THOUGHTFORM_B_FILE_FILE(THOUGHTFORM_B_FILE);
    THOUGHTFORM_B_FILE_FILE << "THOUGHTFORM B";
    std::ofstream AMPLIFIER_FILE_FILE(AMPLIFIER_FILE);
    AMPLIFIER_FILE_FILE << "AMPLIFIER";
    std::ofstream HSUPLINK_FILE_FILE(HSUPLINK_FILE);
    HSUPLINK_FILE_FILE << HOLOLINK_CONTENTS.str();

    std::cout << "Holo-Link files created." << std::endl;
    std::cout << "Remember to create your INTENTIONS.TXT file, in this folder, with all your intentions for the Holo-Link." << std::endl;
    std::cout << "You may do one to a line, or however you feel." << std::endl;
    std::cout << "You may now run with the --usehololink option." << std::endl;
    std::cout << "When using --usehololink, the option --intent, will be ignored, and INTENTIONS.TXT will be used instead." << std::endl;
    std::cout << "Good Luck!" << std::endl;
}

std::string MultiplyStrings(const std::string &num1, const std::string &num2)
{
    int len1 = num1.size();
    int len2 = num2.size();
    std::vector<int> result(len1 + len2, 0);

    for (int i = len1 - 1; i >= 0; --i)
    {
        for (int j = len2 - 1; j >= 0; --j)
        {
            int mul = (num1[i] - '0') * (num2[j] - '0');
            int sum = mul + result[i + j + 1];

            result[i + j + 1] = sum % 10;
            result[i + j] += sum / 10;
        }
    }

    std::string resultStr;
    for (int num : result)
    {
        if (!(resultStr.empty() && num == 0))
        {
            resultStr.push_back(num + '0');
        }
    }

    return resultStr.empty() ? "0" : resultStr;
}

int main(int argc, char **argv)
{
    std::signal(SIGINT, signalHandler);
    std::string intention, process_intention, intention_value, duration, param_duration;
    std::string param_intention, param_intention_2, param_boostlevel, param_color;
    std::string param_usehololink, runtime_formatted, ref_rate;
    std::string suffix_value = "HZ", HSUPLINK_FILE, param_restevery, param_restfor, param_file2 = "X";
    std::string param_compress, param_hashing, useHashing, useCompression, intention_hashed;
    std::string totalIterations = "0", totalFreq = "0", param_file = "X", intention_display = "", loading_message = "LOADING INTO MEMORY...";
    unsigned long long int multiplier = 0;
    unsigned long long int hashMultiplier = 0, freq = 0;
    int seconds = 0, restevery_int = 0, restfor_int = 0;
    int digits = 0, freqDigits = 0;
    float ram_size_value = 1;

    param_duration = "UNTIL STOPPED";
    param_intention = "X";
    param_color = "WHITE";
    param_usehololink = "NO";
    param_boostlevel = "0";
    param_restevery = "0";
    param_restfor = "0";
    param_hashing = "X";
    param_compress = "X";
    HSUPLINK_FILE = "HSUPLINK.TXT";

    for (int i = 1; i < argc; i++)
    {
        if (!strcmp(argv[i], "-h") || !strcmp(argv[i], "--help"))
        {
            print_help();
            std::exit(EXIT_SUCCESS);
        }
        else if (!strcmp(argv[i], "-n") || !strcmp(argv[i], "--colorhelp"))
        {
            print_color_help();
            std::exit(EXIT_SUCCESS);
        }
        else if (!strcmp(argv[i], "-d") || !strcmp(argv[i], "--dur"))
        {
            param_duration = argv[i + 1];
        }
        else if (!strcmp(argv[i], "-m") || !strcmp(argv[i], "--imem"))
        {
            ram_size_value = std::atof(argv[i + 1]);
        }
        else if (!strcmp(argv[i], "-b") || !strcmp(argv[i], "--boostlevel"))
        {
            param_boostlevel = argv[i + 1];
        }
        else if (!strcmp(argv[i], "-p") || !strcmp(argv[i], "--createnestingfiles"))
        {
            create_nesting_files();
            std::exit(EXIT_SUCCESS);
        }
        else if (!strcmp(argv[i], "-i") || !strcmp(argv[i], "--intent"))
        {
            param_intention_2 = argv[i + 1];
            param_intention = (!strcmp(argv[i], "-u") || !strcmp(argv[i], "--usehololink")) ? HSUPLINK_FILE : param_intention_2;
        }
        else if (!strcmp(argv[i], "-u") || !strcmp(argv[i], "--usehololink"))
        {
            param_usehololink = "YES";
        }
        else if (!strcmp(argv[i], "-c") || !strcmp(argv[i], "--color"))
        {
            param_color = argv[i + 1];
            std::transform(param_color.begin(), param_color.end(), param_color.begin(), ::toupper);
        }
        else if (!strcmp(argv[i], "-x") || !strcmp(argv[i], "--createhololinkfiles"))
        {
            create_hololink_files();
            std::exit(EXIT_SUCCESS);
        }
        else if (!strcmp(argv[i], "-s") || !strcmp(argv[i], "--suffix"))
        {
            suffix_value = argv[i + 1];
            std::transform(suffix_value.begin(), suffix_value.end(), suffix_value.begin(), ::toupper);
        }
        else if (!strcmp(argv[i], "-e") || !strcmp(argv[i], "--restevery"))
        {
            param_restevery = argv[i + 1];
            restevery_int = std::stoi(param_restevery);
        }
        else if (!strcmp(argv[i], "-r") || !strcmp(argv[i], "--restfor"))
        {
            param_restfor = argv[i + 1];
            restfor_int = std::stoi(param_restfor);
        }
        else if (!strcmp(argv[i], "-g") || !strcmp(argv[i], "--hashing"))
        {
            param_hashing = argv[i + 1];
            std::transform(param_hashing.begin(), param_hashing.end(), param_hashing.begin(), ::toupper);
        }
        else if (!strcmp(argv[i], "-x") || !strcmp(argv[i], "--compress"))
        {
            param_compress = argv[i + 1];
            std::transform(param_compress.begin(), param_compress.end(), param_compress.begin(), ::toupper);
        }
        else if (!strcmp(argv[i], "-f") || !strcmp(argv[i], "--file"))
        {
            param_file = argv[i + 1];
        }
        else if (!strcmp(argv[i], "-f2") || !strcmp(argv[i], "--file2"))
        {
            param_file2 = argv[i + 1];
        }
    }

    if (param_boostlevel != "0")
    {
        intention = getBoostIntention(param_boostlevel);
        intention_display = "Using Nesting File Quantumly: NEST-" + param_boostlevel + ".TXT with INTENTIONS.TXT";
    }

    if (param_usehololink == "YES")
    {
        std::cout << "Loading HOLO-LINK Files..." << std::flush;
        intention = getHSUPLINKContents();
        intention_display = HSUPLINK_FILE;
    }

    unsigned long long int INTENTION_MULTIPLIER = static_cast<unsigned long long int>(ram_size_value * 1024 * 1024 * 512);
    unsigned long long int free_memory = get_ninety_percent_free_memory();

    if (free_memory != static_cast<unsigned long long>(-1))
    {
        if (free_memory < INTENTION_MULTIPLIER)
        {
            INTENTION_MULTIPLIER = free_memory;
        }
    }
    else
    {
        std::cout << "Error retrieving memory information." << std::endl;
        return 0;
    }

#ifndef _WIN32
    // Set the terminal color based on the --color flag.
    if (param_color == "DEFAULT")
        std::cout << DEFAULT << std::flush;
    else if (param_color == "BLACK")
        std::cout << BLACK << std::flush;
    else if (param_color == "RED")
        std::cout << RED << std::flush;
    else if (param_color == "GREEN")
        std::cout << GREEN << std::flush;
    else if (param_color == "YELLOW")
        std::cout << YELLOW << std::flush;
    else if (param_color == "BLUE")
        std::cout << BLUE << std::flush;
    else if (param_color == "MAGENTA")
        std::cout << MAGENTA << std::flush;
    else if (param_color == "CYAN")
        std::cout << CYAN << std::flush;
    else if (param_color == "LIGHTGRAY")
        std::cout << LIGHTGRAY << std::flush;
    else if (param_color == "DARK_GRAY")
        std::cout << DARKGRAY << std::flush;
    else if (param_color == "LIGHTRED")
        std::cout << LIGHTRED << std::flush;
    else if (param_color == "LIGHTGREEN")
        std::cout << LIGHTGREEN << std::flush;
    else if (param_color == "LIGHTYELLOW")
        std::cout << LIGHTYELLOW << std::flush;
    else if (param_color == "LIGHTBLUE")
        std::cout << LIGHTBLUE << std::flush;
    else if (param_color == "LIGHTMAGENTA")
        std::cout << LIGHTMAGENTA << std::flush;
    else if (param_color == "LIGHTCYAN")
        std::cout << LIGHTCYAN << std::flush;
    else if (param_color == "WHITE")
        std::cout << WHITE << std::flush;
#else
    if (param_color == "BLACK")
        SetConsoleTextAttribute(hConsole, BLACK);
    else if (param_color == "BLUE")
        SetConsoleTextAttribute(hConsole, BLUE);
    else if (param_color == "GREEN")
        SetConsoleTextAttribute(hConsole, GREEN);
    else if (param_color == "CYAN")
        SetConsoleTextAttribute(hConsole, CYAN);
    else if (param_color == "RED")
        SetConsoleTextAttribute(hConsole, RED);
    else if (param_color == "MAGENTA")
        SetConsoleTextAttribute(hConsole, MAGENTA);
    else if (param_color == "LIGHTYELLOW")
        SetConsoleTextAttribute(hConsole, LIGHTYELLOW);
    else if (param_color == "LIGHTGRAY")
        SetConsoleTextAttribute(hConsole, LIGHTGRAY);
    else if (param_color == "DARKGRAY")
        SetConsoleTextAttribute(hConsole, DARKGRAY);
    else if (param_color == "LIGHTBLUE")
        SetConsoleTextAttribute(hConsole, LIGHTBLUE);
    else if (param_color == "LIGHTGREEN")
        SetConsoleTextAttribute(hConsole, LIGHTGREEN);
    else if (param_color == "LIGHTCYAN")
        SetConsoleTextAttribute(hConsole, LIGHTCYAN);
    else if (param_color == "LIGHTRED")
        SetConsoleTextAttribute(hConsole, LIGHTRED);
    else if (param_color == "LIGHTMAGENTA")
        SetConsoleTextAttribute(hConsole, LIGHTMAGENTA);
    else if (param_color == "YELLOW")
        SetConsoleTextAttribute(hConsole, YELLOW);
    else if (param_color == "WHITE")
        SetConsoleTextAttribute(hConsole, WHITE);
#endif // Not Windows
    std::locale comma_locale(std::locale(), new comma_numpunct());
    std::cout.imbue(comma_locale);

    std::cout << "Intention Repeater MAX CUDA v5.23 (c)2020-2024" << std::endl;
    std::cout << "by Anthro Teacher aka Thomas Sweet." << std::endl
              << std::endl;

    std::string file_contents_original, file_contents, file_contents2_original, file_contents2, intention_original;

    if (param_boostlevel == "0" && param_usehololink == "NO")
    {
        if (param_intention == "X" && param_file == "X" && param_file2 == "X")
        {
            while (!interrupted)
            {
                std::cout << "Enter your Intention: ";
                if (!std::getline(std::cin, intention_original))
                {
                    // If getline fails (e.g., due to an interrupt), break out of the loop immediately
                    interrupted.store(true); // Ensure the flag is set if not already
                    return 0;
                }

                //std::cout << "Intention_original: " << intention_original << " intention_original.empty() " << intention_original.empty() << std::endl;

                if (!intention_original.empty())
                {
                    break; // Successfully got an intention, exit the loop
                }
                else if (!interrupted)
                {
                    // Only show the message if we're not interrupted
                    std::cout << "The intention cannot be empty. Please try again.\n";
                }
            }
            //intention_value = intention_original;
        }
        else
        {
            if (param_intention != "X") {
                intention_original = param_intention;
            }
            //intention_value = param_intention;
            intention_display = intention_original;
        }
    }

    if (param_file != "X" && param_boostlevel == "0" && param_usehololink == "NO")
    {
        // Open param_intent file and read the full file contents into intention
        readFileContents(param_file, file_contents_original);
        //intention_display += "Contents of: ";
    }
    if (param_file2 != "X" && param_boostlevel == "0" && param_usehololink == "NO")
    {
        // Open param_intent file and read the full file contents into intention
        readFileContents(param_file2, file_contents2_original);
        //intention_display = "Contents of: ";
    }

    size_t length1 = file_contents_original.size();
    size_t length2 = file_contents2_original.size();
    size_t length3 = intention_original.size();

    size_t max_length = (std::max)({length1, length2, length3});

    if (intention_original != "" && intention_original != "X" && param_boostlevel == "0" && param_usehololink == "NO")
    {
        // Normalize intention
        while (intention.length() < max_length)
        {
            intention += intention_original;
        }
        intention_display = intention_original;
    }

    if (param_file != "X" && param_boostlevel == "0" && param_usehololink == "NO")
    {
        // Normalize file_contents
        while (file_contents.length() < max_length)
        {
            file_contents += file_contents_original;
        }
        intention_display += "(" + param_file + ")";
    }
    if (param_file2 != "X" && param_boostlevel == "0" && param_usehololink == "NO")
    {
        // Normalize file_contents2
        while (file_contents2.length() < max_length)
        {
            file_contents2 += file_contents2_original;
        }
        intention_display += "(" + param_file2 + ")";
    }

    intention += file_contents + file_contents2;

    if (INTENTION_MULTIPLIER > 0)
    {
        // ...

        std::cout << loading_message << std::endl;
        std::string temp = intention;

        while (intention_value.length() < INTENTION_MULTIPLIER)
        {
            intention_value += intention;
            ++multiplier;
        }
        --multiplier; // Account for having to reduce at the end.

        long long int intention_value_length = intention_value.length();
        long long int intention_length = intention.length();
        long long int intention_length_val = intention_value_length - intention_length;

        intention_value = intention_value.substr(0, intention_length_val);
        digits = std::to_string(multiplier).length();
    }

    if (INTENTION_MULTIPLIER == 0)
    {
        intention_value = intention;
        multiplier = 1;
    }

    if (!interrupted && param_hashing == "X")
    {
        std::cout << "Use Hashing (y/N): ";
        if (!std::getline(std::cin, useHashing))
        {
            interrupted.store(true);
            if (interrupted)
            {
                // std::cerr << "Interrupted during hashing input. Exiting configuration.\n";
                return 0;
            }
        }
        transform(useHashing.begin(), useHashing.end(), useHashing.begin(), ::tolower);
    }
    else if (!interrupted)
    {
        useHashing = param_hashing;
        transform(useHashing.begin(), useHashing.end(), useHashing.begin(), ::tolower);
    }

    if (!interrupted && param_compress == "X")
    {
        std::cout << "Use Compression (y/N): ";
        if (!std::getline(std::cin, useCompression))
        {
            interrupted.store(true);
            if (interrupted)
            {
                // std::cerr << "Interrupted during compression input. Exiting configuration.\n";
                return 0;
            }
        }
        transform(useCompression.begin(), useCompression.end(), useCompression.begin(), ::tolower);
    }
    else if (!interrupted)
    {
        useCompression = param_compress;
        transform(useCompression.begin(), useCompression.end(), useCompression.begin(), ::tolower);
    }

    if (multiplier > 0)
    {
        std::cout << "Multiplier: " << display_suffix(std::to_string(multiplier), digits - 1, "Iterations") << std::endl;
    }

    if (useHashing == "y" || useHashing == "yes")
    {
        std::cout << "Hashing...          \r";
        intention_hashed = picosha2::hash256_hex_string(intention_value);
        if (INTENTION_MULTIPLIER > 0)
        {
            intention_value.clear();
            while (intention_value.length() < INTENTION_MULTIPLIER)
            {
                intention_value += intention_hashed;
                ++hashMultiplier;
            }
        }
        else
        {
            intention_value = intention_hashed;
            hashMultiplier = 1;
        }

        digits = std::to_string(hashMultiplier).length();
        std::cout << "Hash Multiplier: " << display_suffix(std::to_string(hashMultiplier), digits - 1, "Iterations") << std::endl;
    }
    else
    {
        hashMultiplier = 1;
    }

    long long int originalIntentionSize, compressedIntentionSize, compressionFactor;
    int compressionFactor_digits, compressedIntentionSize_digits, originalIntention_digits;

    if (useCompression == "y" || useCompression == "yes")
    {
        std::cout << "Compressing...          \r";
        originalIntentionSize = intention_value.length() * 2;
        intention_value = compressMessage(intention_value);
        compressedIntentionSize = intention_value.length() * 2;
        compressionFactor = (originalIntentionSize / compressedIntentionSize);

        compressionFactor_digits = std::to_string(compressionFactor).length();
        compressedIntentionSize_digits = std::to_string(compressedIntentionSize).length();
        originalIntention_digits = std::to_string(originalIntentionSize).length();

        std::cout << "Compression: " << display_suffix(std::to_string(compressionFactor), compressionFactor_digits - 1, "Iterations") << "X ["
                  << display_suffix(std::to_string(originalIntentionSize), originalIntention_digits - 1, "Frequency") << "B -> "
                  << display_suffix(std::to_string(compressedIntentionSize), compressedIntentionSize_digits - 1, "Frequency") << "B]     " << std::endl;
    }

    // Allocate memory on the GPU for intentionMultiplied and freq
    char *d_intentionMultiplied;
    unsigned long long int *d_freq;
    hipMalloc(&d_intentionMultiplied, intention_value.size());
    hipMalloc(&d_freq, sizeof(unsigned long long int));

    // Copy intentionMultiplied to the GPU
    hipMemcpy(d_intentionMultiplied, intention_value.c_str(), intention_value.size(), hipMemcpyHostToDevice);

    duration = param_duration;

    process_intention.reserve(intention_value.size() + 20);

    auto start = std::chrono::high_resolution_clock::now();
    auto end = std::chrono::high_resolution_clock::now();

    do
    {
        start = std::chrono::high_resolution_clock::now();
        end = std::chrono::high_resolution_clock::now();
        // Set freq to 0 on the GPU
        hipMemset(d_freq, 0, sizeof(unsigned long long int));

        while ((std::chrono::duration_cast<std::chrono::seconds>(end - start).count() < 1))
        {
            // Launch the CUDA kernel for intention repeating and frequency updating
            int blockSize = 256;
            int numBlocks = (intention_value.size() + blockSize - 1) / blockSize;
            intentionRepeaterKernel<<<numBlocks, blockSize>>>(d_intentionMultiplied, d_freq, intention_value.size());

            // Wait for the GPU to finish before accessing on host
            hipDeviceSynchronize();

            // Copy the updated freq back to the CPU
            hipMemcpy(&freq, d_freq, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
            end = std::chrono::high_resolution_clock::now();
        }
        ++seconds;

        totalFreq = MultiplyStrings(std::to_string(freq), std::to_string(multiplier));
        totalFreq = MultiplyStrings(totalFreq, std::to_string(hashMultiplier));
        totalIterations = FindSum(totalIterations, totalFreq);
        digits = totalIterations.length();
        freqDigits = totalFreq.length();
        freq = 0;

        runtime_formatted = FormatTimeRun(seconds);

        if (suffix_value == "EXP")
        {
            std::cout << "[" + runtime_formatted + "]"
                      << " (" << std::setprecision(3) << std::fixed
                      << (std::stoull(totalIterations.substr(0, 4)) / std::pow(10, 3)) << "x10^" << digits - 1 << " / "
                      << (std::stoull(totalFreq.substr(0, 4)) / std::pow(10, 3)) << "x10^" << freqDigits - 1
                      << " Hz): " << intention_display << "     \r" << std::flush;
        }
        else // suffix_value = "HZ"
        {
            std::cout << "[" + runtime_formatted + "]"
                      << " (" << display_suffix(totalIterations, digits - 1, "Iterations") << " / "
                      << display_suffix(totalFreq, freqDigits - 1, "Frequency")
                      << "Hz): " << intention_display << "     \r" << std::flush;
        }

        if (runtime_formatted == duration || interrupted)
        {
            std::cout << std::endl
                      << std::flush;
            std::exit(EXIT_SUCCESS);
        }

        if ((restevery_int > 0) && (seconds % restevery_int == 0))
        {
            start = std::chrono::high_resolution_clock::now();
            end = std::chrono::high_resolution_clock::now();

            if (suffix_value == "EXP")
            {
                std::cout << "[" + runtime_formatted + "]"
                          << " (" << std::setprecision(3) << std::fixed
                          << (std::stoull(totalIterations.substr(0, 4)) / std::pow(10, 3)) << "x10^" << digits - 1 << " / "
                          << (std::stoull(totalFreq.substr(0, 4)) / std::pow(10, 3)) << "x10^" << freqDigits - 1
                          << " Hz): " << intention_display << "     \r" << std::flush;
            }
            else // suffix_value = "HZ"
            {
                std::cout << "[" + runtime_formatted + "]"
                          << " (" << display_suffix(totalIterations, digits - 1, "Iterations") << " / "
                          << display_suffix(totalFreq, freqDigits - 1, "Frequency")
                          << "Hz): " << intention_display << "     \r" << std::flush;
            }
            while ((std::chrono::duration_cast<std::chrono::seconds>(end - start).count() < restfor_int))
            {
                end = std::chrono::high_resolution_clock::now();
            }
        }
    } while (1);

    hipFree(d_intentionMultiplied);
    hipFree(d_freq);

#ifndef _WIN32
    std::cout << WHITE << std::flush;
#else
    SetConsoleTextAttribute(hConsole, WHITE);
#endif

    return 0;
}

/*
    Intention Repeater MAX CUDA v1.0 created by Thomas Sweet.
	CUDA functions by Karteek Sheri.
    Created 11/2/2020 for C++.
	Requires: CUDA Toolkit: https://developer.nvidia.com/cuda-toolkit
	Requires: Visual Studio 2019 Community for C++: https://visualstudio.microsoft.com/downloads/
	Requires: Add location of cl.exe to Windows PATH.
	To compile: nvcc intention_repeater_max_cuda.cu -O 3 -o intention_repeater_max_cuda.exe
    Repeats your intention up to 500+ million times per second to make things happen.
    When compiled, this is more powerful than the Python version.
    Intention Repeater MAX is powered by a Servitor (20 Years / 2000+ hours in the making) [HR 6819 Black Hole System].
    Servitor Info: https://enlightenedstates.com/2017/04/07/servitor-just-powerful-spiritual-tool/
    Website: https://www.intentionrepeater.com/
    Forum: https://forums.intentionrepeater.com/
    Licensed under GNU General Public License v3.0
    This means you can modify, redistribute and even sell your own modified software, as long as it's open source too and released under this same license.
    https://choosealicense.com/licenses/gpl-3.0/
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

#include <string>

#include <string.h>

#include <math.h>

#include <iostream>

#include <time.h>

#include <ctime>

#include <ratio>

#include <chrono>

#include <iomanip>

#include <locale.h>

#define USEGPU

using namespace std;
using namespace std::chrono;

std::string FormatTimeRun(int seconds_elapsed);

#define ONE_MINUTE 60
#define ONE_HOUR 3600
#define PROCESS_STATEMENT " REGULATE AND INTEGRATE. IT IS DONE. OM."

#ifdef USEGPU
    //CUDA code added by Karteek Sheri
    __device__ __managed__ unsigned long long int iterations = 0, frequency_count = 0;

    __global__ void intention_on_gpu(char *device_intention_value_array, const int num_chars){

        for(int i=0; i<num_chars;i++){
            device_intention_value_array[i] = device_intention_value_array[i];}
        atomicAdd(&iterations,1);
        atomicAdd(&frequency_count,1);

    }

#endif //end of CUDA code

class comma_numpunct: public std::numpunct < char > {
    protected: virtual char do_thousands_sep() const {
        return ',';
    }

    virtual std::string do_grouping() const {
        return "\03";
    }
};

static const char* short_scale[] = {
  "",
  "k",
  "M",
  "B",
  "T",
  "Q"
};

static const char* short_scale_hz[] = {
  "",
  "k",
  "M",
  "G",
  "T",
  "P"
};

const char* scale(double n, int decimals = 1, const char* units[] = short_scale)
{
  /*
   * Number of digits in n is given by
   * 10^x = n ==> x = log(n)/log(10) = log_10(n).
   *
   * So 1000 would be 1 + floor(log_10(10^3)) = 4 digits.
   */
  int digits = n==0? 0 : 1+floor(log10l(fabs(n)));

  // determine base 10 exponential
  int exp = digits <= 4? 0 : 3*((digits-1)/3);

  // normalized number
  double m = n / powl(10, exp);

  // no decimals? then don't print any
  if ( m - static_cast<long>(n) == 0 )
    decimals = 0;

  // don't print unit for exp<3
  static char s[32];
  static const char* fmt[] = {"%1.*lf%s", "%1.*lf"};
  sprintf(s, fmt[exp<3], decimals, m, units[exp/3]);
  return s;
}

const char* suffix(double n, int decimals = 1)
{
  static char s[32];
  strcpy(s, scale(n, decimals, short_scale));
  return s;
}

const char* suffix_hz(double n, int decimals = 1)
{
  static char s[32];
  strcpy(s, scale(n, decimals, short_scale_hz));

  return s;
}

int main(int argc, char ** argv) {
    std::string intention, intention_value, process_intention, duration, param_duration, param_intention, runtime_formatted;
    #ifndef USEGPU
        unsigned long long int iterations = 0, frequency_count = 0;
    #endif
    int seconds = 0;

    if (argc == 3) {
        param_duration = argv[1];
        param_intention = argv[2];

    } else {
        param_duration = "Until Stopped";
        param_intention = "";
    }

    std::locale comma_locale(std::locale(), new comma_numpunct());
    std::cout.imbue(comma_locale);

    cout << "Intention Repeater MAX CUDA v1.0 created by Thomas Sweet." << endl;
	cout << "CUDA functions by Karteek Sheri." << endl;
    cout << "This software comes with no guarantees or warranty of any kind and is for entertainment purposes only." << endl;
    cout << "Press Ctrl-C to quit." << endl << endl;

    if ((param_intention) == "") {
        cout << "Intention: ";
        std::getline(std::cin, intention);

    } else {
        intention = param_intention;
    }

    intention_value = intention + PROCESS_STATEMENT;
#ifdef USEGPU
    const int num_chars = intention_value.length();
	// declaring character array
    char* intention_value_array;
    intention_value_array = (char *)malloc((num_chars+1) * sizeof(char));
    //char device_intention_array[n+1];
    char *device_intention_value_array;
    strcpy(intention_value_array, intention_value.c_str());

	//for (int i = 0; i < num_chars; i++)
	//	cout << intention_value_array[i];
  
    hipHostAlloc((void**)&device_intention_value_array,(num_chars+1)*sizeof(char),hipHostMallocDefault);
    hipMemcpy(device_intention_value_array,intention_value_array,(num_chars+1)*sizeof(char),hipMemcpyHostToDevice);
#endif


    duration = param_duration;

    auto start = std::chrono::system_clock::now();
    auto end = std::chrono::system_clock::now();

    do {
        start = std::chrono::system_clock::now();
		frequency_count = 0;
        while ((std::chrono::duration_cast < std::chrono::seconds > (end - start).count() != 1)) {
    #ifndef USEGPU
            process_intention = intention_value; //This is the Intention Repeater call that actually does the work with the Servitor.
            iterations += 1;
            frequency_count += 1;
    #else
            intention_on_gpu<<<65535,1024>>>(device_intention_value_array, num_chars);
            hipDeviceSynchronize();
    #endif
            end = std::chrono::system_clock::now();
        }
        seconds += 1;
        runtime_formatted = FormatTimeRun(seconds);
        std::cout << "[" + runtime_formatted + "]" << " (" << suffix(iterations) << "/" << suffix_hz(frequency_count) << "Hz): " << intention << "     \r" << std::flush;
        if (runtime_formatted == duration) {
			std::cout << endl << std::flush;
            exit(0);
        }
    } while (1);
	std::cout << endl << "[" + runtime_formatted + "]" << " (" << suffix(iterations) << "|" << suffix_hz(frequency_count) << "Hz): " << intention << "     " << endl << std::flush;
    
	#ifdef USEGPU
		hipFree(device_intention_value_array);
	#endif

    return 0;
}

std::string FormatTimeRun(int seconds_elapsed) {
    int hour, min, sec;

    std::string hour_formatted, min_formatted, sec_formatted;

    hour = seconds_elapsed / ONE_HOUR;
    seconds_elapsed -= hour * ONE_HOUR;
    min = seconds_elapsed / ONE_MINUTE;
    seconds_elapsed -= min * ONE_MINUTE;
    sec = seconds_elapsed;

    if (hour < 10) {
        hour_formatted = "0" + std::to_string(hour);
    } else {
        hour_formatted = std::to_string(hour);
    }

    if (min < 10) {
        min_formatted = "0" + std::to_string(min);
    } else {
        min_formatted = std::to_string(min);
    }

    if (sec < 10) {
        sec_formatted = "0" + std::to_string(sec);
    } else {
        sec_formatted = std::to_string(sec);
    }

    return hour_formatted + ":" + min_formatted + ":" + sec_formatted;
}